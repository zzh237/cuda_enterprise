#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

__global__ void sobel_filter(unsigned char* input, unsigned char* output, int width, int height) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x > 0 && y > 0 && x < width - 1 && y < height - 1) {
        float gx = input[(y-1)*width + (x-1)] - input[(y-1)*width + (x+1)] 
                 + 2 * input[y*width + (x-1)] - 2 * input[y*width + (x+1)] 
                 + input[(y+1)*width + (x-1)] - input[(y+1)*width + (x+1)];

        float gy = input[(y-1)*width + (x-1)] + 2 * input[(y-1)*width + x] + input[(y-1)*width + (x+1)] 
                 - input[(y+1)*width + (x-1)] - 2 * input[(y+1)*width + x] - input[(y+1)*width + (x+1)];

        output[y*width + x] = sqrt(gx*gx + gy*gy);
    }
}

int main(int argc, char** argv) {
    if(argc != 3) {
        std::cerr << "Usage: ./sobel_filter <input_image_path> <output_image_path>" << std::endl;
        return -1;
    }

    cv::Mat image = cv::imread(argv[1], cv::IMREAD_GRAYSCALE);
    if(!image.data) {
        std::cerr << "Error reading image!" << std::endl;
        return -1;
    }

    cv::Mat output(image.rows, image.cols, CV_8UC1);

    unsigned char* d_input;
    unsigned char* d_output;

    hipMalloc(&d_input, image.rows * image.cols);
    hipMalloc(&d_output, image.rows * image.cols);

    hipMemcpy(d_input, image.data, image.rows * image.cols, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((image.cols + threadsPerBlock.x - 1) / threadsPerBlock.x, (image.rows + threadsPerBlock.y - 1) / threadsPerBlock.y);
    sobel_filter<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, image.cols, image.rows);

    hipMemcpy(output.data, d_output, image.rows * image.cols, hipMemcpyDeviceToHost);

    cv::imwrite(argv[2], output);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
